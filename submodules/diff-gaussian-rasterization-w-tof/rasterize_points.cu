#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include <fstream>
#include <string>
#include <functional>

std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

std::tuple<int, 
torch::Tensor, torch::Tensor, // Color, Phasor
torch::Tensor, torch::Tensor, torch::Tensor, // Depth, Normal, Acc
torch::Tensor, torch::Tensor, torch::Tensor, // Entropy, Depth Distortion, Amplitude Distortion
torch::Tensor, // Pixels
torch::Tensor, // Distributions
torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor> 
RasterizeGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& colors, const torch::Tensor& phasors,
    const torch::Tensor& opacity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx, 
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const torch::Tensor& sh, const torch::Tensor& sh_p,
	const int degree,
	const torch::Tensor& campos,
	const bool prefiltered,
	const bool debug,
	const float near_n,
	const float far_n,
	const float depth_range,
	const bool use_view_dependent_phase,
	const float phase_offset, const float dc_offset
	)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }
  
  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);

  torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  torch::Tensor out_phasor = torch::full({NUM_CHANNELS_PHASOR, H, W}, 0.0, float_opts);
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  torch::Tensor pixels = torch::zeros({P, 1}, means3D.options());

  // Additional images to render
  torch::Tensor out_depth = torch::full({1, H, W}, 0.0, float_opts);
  torch::Tensor out_normal = torch::full({3, H, W}, 0.0, float_opts);
  torch::Tensor out_acc = torch::full({1, H, W}, 0.0, float_opts);
  torch::Tensor out_entropy = torch::full({1, H, W}, 0.0, float_opts);
  torch::Tensor out_depth_distortion = torch::full({1, H, W}, 0.0, float_opts);
  torch::Tensor out_amp_distortion = torch::full({1, H, W}, 0.0, float_opts);
  torch::Tensor out_distribution = torch::full({NUM_SAMPLES_EACH_RAY * 3, H, W}, 0.0, float_opts);

  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);
  
  int rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if (sh.defined()) 
	  {
		if(sh.size(0) != 0)
		{
			M = sh.size(1); // max no of sh coefficient --> 1, 4, 9, 16
		}
	  }

	  int M_p = 0;
	  if (sh_p.defined()) 
	  {
		if(sh_p.size(0) != 0)
		{
			M_p = sh_p.size(1);
		}
	  }

	  rendered = CudaRasterizer::Rasterizer::forward(
	    geomFunc,
		binningFunc,
		imgFunc,
	    P, degree, M, M_p,
		background.contiguous().data<float>(),
		W, H,
		means3D.contiguous().data<float>(),
		sh.contiguous().data_ptr<float>(), sh_p.contiguous().data_ptr<float>(),
		colors.contiguous().data<float>(), phasors.contiguous().data<float>(), 
		opacity.contiguous().data<float>(), 
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		out_color.contiguous().data<float>(), out_phasor.contiguous().data<float>(), 
		out_depth.contiguous().data<float>(), out_normal.contiguous().data<float>(), out_acc.contiguous().data<float>(), 
		out_entropy.contiguous().data<float>(), out_depth_distortion.contiguous().data<float>(), out_amp_distortion.contiguous().data<float>(),
		pixels.contiguous().data<float>(),
		out_distribution.contiguous().data<float>(),
		radii.contiguous().data<int>(),
		debug, 
		near_n, far_n, depth_range, 
		use_view_dependent_phase,
		phase_offset, dc_offset
		);
  }
  return std::make_tuple(
	rendered, 
	out_color, out_phasor, 
	out_depth, out_normal, out_acc, 
	out_entropy, out_depth_distortion, out_amp_distortion, 
	pixels, 
	out_distribution,
	radii, geomBuffer, binningBuffer, imgBuffer);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
RasterizeGaussiansBackwardCUDA(
 	const torch::Tensor& background,
	const torch::Tensor& means3D,
	const torch::Tensor& radii,
    const torch::Tensor& colors, const torch::Tensor& phasors,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
	const float tan_fovx,
	const float tan_fovy,
    const torch::Tensor& dL_dout_color, const torch::Tensor& dL_dout_phasor, 
	const torch::Tensor& dL_dout_depth, const torch::Tensor& dL_dout_normal, const torch::Tensor& dL_dout_acc, 
	const torch::Tensor& dL_dout_entropy, const torch::Tensor& dL_dout_depth_distortion, const torch::Tensor& dL_dout_amp_distortion,
	const torch::Tensor& sh, const torch::Tensor& sh_p,
	const int degree,
	const torch::Tensor& campos,
	const torch::Tensor& geomBuffer,
	const int R,
	const torch::Tensor& binningBuffer,
	const torch::Tensor& imageBuffer,
	const bool debug,
	const float near_n,
	const float far_n,
	const float depth_range,
	const bool use_view_dependent_phase,
	const float phase_offset,
	const float dc_offset
	) 
{
  const int P = means3D.size(0);
  const int H = dL_dout_color.size(1);
  const int W = dL_dout_color.size(2);
  
  int M = 0;
  if (sh.defined())
  {
	if(sh.size(0) != 0)
	{	
		M = sh.size(1);
	}
  }

  int M_p = 0;
  if (sh_p.defined())
  {
	if(sh_p.size(0) != 0)
	{	
		M_p = sh_p.size(1);
	}
  }

  torch::Tensor dL_dmeans3D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dmeans2D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dcolors = torch::zeros({P, NUM_CHANNELS}, means3D.options());
  torch::Tensor dL_dphasors = torch::zeros({P, NUM_CHANNELS_PHASOR}, means3D.options());
  torch::Tensor dL_ddists_to_light = torch::zeros({P, 1}, means3D.options()); // For depth loss. View space
  torch::Tensor dL_dzs_ndc = torch::zeros({P, 1}, means3D.options()); // For depth distortion loss.
  torch::Tensor dL_dconic = torch::zeros({P, 2, 2}, means3D.options());
  torch::Tensor dL_dopacity = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dcov3D = torch::zeros({P, 6}, means3D.options());
  torch::Tensor dL_dsh = torch::zeros({P, M, NUM_CHANNELS}, means3D.options());
  torch::Tensor dL_dsh_p = torch::zeros({P, M_p, NUM_CHANNELS_CWTOF}, means3D.options()); // phase & amp
  torch::Tensor dL_dscales = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());
  torch::Tensor dL_dphase_offset = torch::zeros({1}, means3D.options());
  torch::Tensor dL_ddc_offset = torch::zeros({1}, means3D.options());
  
  if(P != 0)
  {  
	  CudaRasterizer::Rasterizer::backward(P, degree, M, M_p, R,
		background.contiguous().data<float>(),
		W, H, 
		means3D.contiguous().data<float>(),
		sh.contiguous().data<float>(), sh_p.contiguous().data<float>(),
		colors.contiguous().data<float>(), phasors.contiguous().data<float>(),
		scales.data_ptr<float>(),
		scale_modifier,
		rotations.data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(),
		viewmatrix.contiguous().data<float>(),
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		radii.contiguous().data<int>(),
		reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
		reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
		reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
		dL_dout_color.contiguous().data<float>(), dL_dout_phasor.contiguous().data<float>(), 
		dL_dout_depth.contiguous().data<float>(), dL_dout_normal.contiguous().data<float>(), dL_dout_acc.contiguous().data<float>(),
		dL_dout_entropy.contiguous().data<float>(), dL_dout_depth_distortion.contiguous().data<float>(), dL_dout_amp_distortion.contiguous().data<float>(),
		dL_dmeans2D.contiguous().data<float>(),
		dL_dconic.contiguous().data<float>(),  
		dL_dopacity.contiguous().data<float>(),
		dL_dcolors.contiguous().data<float>(), dL_dphasors.contiguous().data<float>(), dL_ddists_to_light.contiguous().data<float>(), dL_dzs_ndc.contiguous().data<float>(),
		dL_dmeans3D.contiguous().data<float>(),
		dL_dcov3D.contiguous().data<float>(),
		dL_dsh.contiguous().data<float>(), dL_dsh_p.contiguous().data<float>(),
		dL_dscales.contiguous().data<float>(),
		dL_drotations.contiguous().data<float>(),
		dL_dphase_offset.contiguous().data<float>(),
		dL_ddc_offset.contiguous().data<float>(),
		debug, 
		near_n, far_n, depth_range, 
		use_view_dependent_phase,
		phase_offset, dc_offset
		);
  }

  return std::make_tuple(dL_dmeans2D, dL_dcolors, dL_dphasors, dL_dopacity, dL_dmeans3D, dL_dcov3D, dL_dsh, dL_dsh_p, dL_dscales, dL_drotations, dL_dphase_offset, dL_ddc_offset);
}

torch::Tensor markVisible(
		torch::Tensor& means3D,
		torch::Tensor& viewmatrix,
		torch::Tensor& projmatrix,
		float znear, float zfar)
{ 
  const int P = means3D.size(0);
  
  torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));
 
  if(P != 0)
  {
	CudaRasterizer::Rasterizer::markVisible(P,
		means3D.contiguous().data<float>(),
		viewmatrix.contiguous().data<float>(),
		projmatrix.contiguous().data<float>(),
		present.contiguous().data<bool>(),
		znear, zfar);
  }
  
  return present;
}