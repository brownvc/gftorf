#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "backward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Backward pass for conversion of spherical harmonics to RGB for
// each Gaussian.
__device__ void computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, const bool* clamped, const glm::vec3* dL_dcolor, glm::vec3* dL_dmeans, glm::vec3* dL_dshs)
{
	// Compute intermediate values, as it is done during forward
	glm::vec3 pos = means[idx];
	glm::vec3 dir_orig = pos - campos;
	glm::vec3 dir = dir_orig / glm::length(dir_orig);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;

	// Use PyTorch rule for clamping: if clamping was applied,
	// gradient becomes 0.
	glm::vec3 dL_dRGB = dL_dcolor[idx];
	dL_dRGB.x *= clamped[3 * idx + 0] ? 0 : 1;
	dL_dRGB.y *= clamped[3 * idx + 1] ? 0 : 1;
	dL_dRGB.z *= clamped[3 * idx + 2] ? 0 : 1;

	glm::vec3 dRGBdx(0, 0, 0);
	glm::vec3 dRGBdy(0, 0, 0);
	glm::vec3 dRGBdz(0, 0, 0);
	float x = dir.x;
	float y = dir.y;
	float z = dir.z;

	// Target location for this Gaussian to write SH gradients to
	glm::vec3* dL_dsh = dL_dshs + idx * max_coeffs;

	// No tricks here, just high school-level calculus.
	float dRGBdsh0 = SH_C0;
	dL_dsh[0] = dRGBdsh0 * dL_dRGB;
	if (deg > 0)
	{
		float dRGBdsh1 = -SH_C1 * y;
		float dRGBdsh2 = SH_C1 * z;
		float dRGBdsh3 = -SH_C1 * x;
		dL_dsh[1] = dRGBdsh1 * dL_dRGB;
		dL_dsh[2] = dRGBdsh2 * dL_dRGB;
		dL_dsh[3] = dRGBdsh3 * dL_dRGB;

		dRGBdx = -SH_C1 * sh[3];
		dRGBdy = -SH_C1 * sh[1];
		dRGBdz = SH_C1 * sh[2];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;

			float dRGBdsh4 = SH_C2[0] * xy;
			float dRGBdsh5 = SH_C2[1] * yz;
			float dRGBdsh6 = SH_C2[2] * (2.f * zz - xx - yy);
			float dRGBdsh7 = SH_C2[3] * xz;
			float dRGBdsh8 = SH_C2[4] * (xx - yy);
			dL_dsh[4] = dRGBdsh4 * dL_dRGB;
			dL_dsh[5] = dRGBdsh5 * dL_dRGB;
			dL_dsh[6] = dRGBdsh6 * dL_dRGB;
			dL_dsh[7] = dRGBdsh7 * dL_dRGB;
			dL_dsh[8] = dRGBdsh8 * dL_dRGB;

			dRGBdx += SH_C2[0] * y * sh[4] + SH_C2[2] * 2.f * -x * sh[6] + SH_C2[3] * z * sh[7] + SH_C2[4] * 2.f * x * sh[8];
			dRGBdy += SH_C2[0] * x * sh[4] + SH_C2[1] * z * sh[5] + SH_C2[2] * 2.f * -y * sh[6] + SH_C2[4] * 2.f * -y * sh[8];
			dRGBdz += SH_C2[1] * y * sh[5] + SH_C2[2] * 2.f * 2.f * z * sh[6] + SH_C2[3] * x * sh[7];

			if (deg > 2)
			{
				float dRGBdsh9 = SH_C3[0] * y * (3.f * xx - yy);
				float dRGBdsh10 = SH_C3[1] * xy * z;
				float dRGBdsh11 = SH_C3[2] * y * (4.f * zz - xx - yy);
				float dRGBdsh12 = SH_C3[3] * z * (2.f * zz - 3.f * xx - 3.f * yy);
				float dRGBdsh13 = SH_C3[4] * x * (4.f * zz - xx - yy);
				float dRGBdsh14 = SH_C3[5] * z * (xx - yy);
				float dRGBdsh15 = SH_C3[6] * x * (xx - 3.f * yy);
				dL_dsh[9] = dRGBdsh9 * dL_dRGB;
				dL_dsh[10] = dRGBdsh10 * dL_dRGB;
				dL_dsh[11] = dRGBdsh11 * dL_dRGB;
				dL_dsh[12] = dRGBdsh12 * dL_dRGB;
				dL_dsh[13] = dRGBdsh13 * dL_dRGB;
				dL_dsh[14] = dRGBdsh14 * dL_dRGB;
				dL_dsh[15] = dRGBdsh15 * dL_dRGB;

				dRGBdx += (
					SH_C3[0] * sh[9] * 3.f * 2.f * xy +
					SH_C3[1] * sh[10] * yz +
					SH_C3[2] * sh[11] * -2.f * xy +
					SH_C3[3] * sh[12] * -3.f * 2.f * xz +
					SH_C3[4] * sh[13] * (-3.f * xx + 4.f * zz - yy) +
					SH_C3[5] * sh[14] * 2.f * xz +
					SH_C3[6] * sh[15] * 3.f * (xx - yy));

				dRGBdy += (
					SH_C3[0] * sh[9] * 3.f * (xx - yy) +
					SH_C3[1] * sh[10] * xz +
					SH_C3[2] * sh[11] * (-3.f * yy + 4.f * zz - xx) +
					SH_C3[3] * sh[12] * -3.f * 2.f * yz +
					SH_C3[4] * sh[13] * -2.f * xy +
					SH_C3[5] * sh[14] * -2.f * yz +
					SH_C3[6] * sh[15] * -3.f * 2.f * xy);

				dRGBdz += (
					SH_C3[1] * sh[10] * xy +
					SH_C3[2] * sh[11] * 4.f * 2.f * yz +
					SH_C3[3] * sh[12] * 3.f * (2.f * zz - xx - yy) +
					SH_C3[4] * sh[13] * 4.f * 2.f * xz +
					SH_C3[5] * sh[14] * (xx - yy));
			}
		}
	}

	// The view direction is an input to the computation. View direction
	// is influenced by the Gaussian's mean, so SHs gradients
	// must propagate back into 3D position.
	glm::vec3 dL_ddir(glm::dot(dRGBdx, dL_dRGB), glm::dot(dRGBdy, dL_dRGB), glm::dot(dRGBdz, dL_dRGB));

	// Account for normalization of direction
	float3 dL_dmean = dnormvdv(float3{ dir_orig.x, dir_orig.y, dir_orig.z }, float3{ dL_ddir.x, dL_ddir.y, dL_ddir.z });

	// Gradients of loss w.r.t. Gaussian means, but only the portion 
	// that is caused because the mean affects the view-dependent color.
	// Additional mean gradient is accumulated in below methods.
	dL_dmeans[idx] += glm::vec3(dL_dmean.x, dL_dmean.y, dL_dmean.z);
}

// Backward pass for conversion of spherical harmonics to phasor for
// each Gaussian.
__device__ void computePhasorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs_p, const bool* clamped_p, const glm::vec2* dL_dcwtof, glm::vec3* dL_dmeans, glm::vec2* dL_dshs_p)
{
	// Compute intermediate values, as it is done during forward
	glm::vec3 pos = means[idx];
	glm::vec3 dir_orig = pos - campos;
	glm::vec3 dir = dir_orig / glm::length(dir_orig);

	glm::vec2* sh = ((glm::vec2*)shs_p) + idx * max_coeffs;

	// Use PyTorch rule for clamping: if clamping was applied,
	// gradient becomes 0.
	glm::vec2 dL_dPA = dL_dcwtof[idx]; // P = phase, A = amplitude
	dL_dPA.y *= clamped_p[idx] ? 0 : 1;

	glm::vec2 dPAdx(0, 0);
	glm::vec2 dPAdy(0, 0);
	glm::vec2 dPAdz(0, 0);
	float x = dir.x;
	float y = dir.y;
	float z = dir.z;

	// Target location for this Gaussian to write SH gradients to
	glm::vec2* dL_dsh_p = dL_dshs_p + idx * max_coeffs;

	// No tricks here, just high school-level calculus.
	float dPAdsh0 = SH_C0;
	dL_dsh_p[0] = dPAdsh0 * dL_dPA;
	if (deg > 0)
	{
		float dPAdsh1 = -SH_C1 * y;
		float dPAdsh2 = SH_C1 * z;
		float dPAdsh3 = -SH_C1 * x;
		dL_dsh_p[1] = dPAdsh1 * dL_dPA;
		dL_dsh_p[2] = dPAdsh2 * dL_dPA;
		dL_dsh_p[3] = dPAdsh3 * dL_dPA;

		dPAdx = -SH_C1 * sh[3];
		dPAdy = -SH_C1 * sh[1];
		dPAdz = SH_C1 * sh[2];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;

			float dPAdsh4 = SH_C2[0] * xy;
			float dPAdsh5 = SH_C2[1] * yz;
			float dPAdsh6 = SH_C2[2] * (2.f * zz - xx - yy);
			float dPAdsh7 = SH_C2[3] * xz;
			float dPAdsh8 = SH_C2[4] * (xx - yy);
			dL_dsh_p[4] = dPAdsh4 * dL_dPA;
			dL_dsh_p[5] = dPAdsh5 * dL_dPA;
			dL_dsh_p[6] = dPAdsh6 * dL_dPA;
			dL_dsh_p[7] = dPAdsh7 * dL_dPA;
			dL_dsh_p[8] = dPAdsh8 * dL_dPA;

			dPAdx += SH_C2[0] * y * sh[4] + SH_C2[2] * 2.f * -x * sh[6] + SH_C2[3] * z * sh[7] + SH_C2[4] * 2.f * x * sh[8];
			dPAdy += SH_C2[0] * x * sh[4] + SH_C2[1] * z * sh[5] + SH_C2[2] * 2.f * -y * sh[6] + SH_C2[4] * 2.f * -y * sh[8];
			dPAdz += SH_C2[1] * y * sh[5] + SH_C2[2] * 2.f * 2.f * z * sh[6] + SH_C2[3] * x * sh[7];

			if (deg > 2)
			{
				float dPAdsh9 = SH_C3[0] * y * (3.f * xx - yy);
				float dPAdsh10 = SH_C3[1] * xy * z;
				float dPAdsh11 = SH_C3[2] * y * (4.f * zz - xx - yy);
				float dPAdsh12 = SH_C3[3] * z * (2.f * zz - 3.f * xx - 3.f * yy);
				float dPAdsh13 = SH_C3[4] * x * (4.f * zz - xx - yy);
				float dPAdsh14 = SH_C3[5] * z * (xx - yy);
				float dPAdsh15 = SH_C3[6] * x * (xx - 3.f * yy);
				dL_dsh_p[9] = dPAdsh9 * dL_dPA;
				dL_dsh_p[10] = dPAdsh10 * dL_dPA;
				dL_dsh_p[11] = dPAdsh11 * dL_dPA;
				dL_dsh_p[12] = dPAdsh12 * dL_dPA;
				dL_dsh_p[13] = dPAdsh13 * dL_dPA;
				dL_dsh_p[14] = dPAdsh14 * dL_dPA;
				dL_dsh_p[15] = dPAdsh15 * dL_dPA;

				dPAdx += (
					SH_C3[0] * sh[9] * 3.f * 2.f * xy +
					SH_C3[1] * sh[10] * yz +
					SH_C3[2] * sh[11] * -2.f * xy +
					SH_C3[3] * sh[12] * -3.f * 2.f * xz +
					SH_C3[4] * sh[13] * (-3.f * xx + 4.f * zz - yy) +
					SH_C3[5] * sh[14] * 2.f * xz +
					SH_C3[6] * sh[15] * 3.f * (xx - yy));

				dPAdy += (
					SH_C3[0] * sh[9] * 3.f * (xx - yy) +
					SH_C3[1] * sh[10] * xz +
					SH_C3[2] * sh[11] * (-3.f * yy + 4.f * zz - xx) +
					SH_C3[3] * sh[12] * -3.f * 2.f * yz +
					SH_C3[4] * sh[13] * -2.f * xy +
					SH_C3[5] * sh[14] * -2.f * yz +
					SH_C3[6] * sh[15] * -3.f * 2.f * xy);

				dPAdz += (
					SH_C3[1] * sh[10] * xy +
					SH_C3[2] * sh[11] * 4.f * 2.f * yz +
					SH_C3[3] * sh[12] * 3.f * (2.f * zz - xx - yy) +
					SH_C3[4] * sh[13] * 4.f * 2.f * xz +
					SH_C3[5] * sh[14] * (xx - yy));
			}
		}
	}

	// The view direction is an input to the computation. View direction
	// is influenced by the Gaussian's mean, so SHs gradients
	// must propagate back into 3D position.
	glm::vec3 dL_ddir(glm::dot(dPAdx, dL_dPA), glm::dot(dPAdy, dL_dPA), glm::dot(dPAdz, dL_dPA));

	// Account for normalization of direction
	float3 dL_dmean = dnormvdv(float3{ dir_orig.x, dir_orig.y, dir_orig.z }, float3{ dL_ddir.x, dL_ddir.y, dL_ddir.z });

	// Gradients of loss w.r.t. Gaussian means, but only the portion 
	// that is caused because the mean affects the view-dependent color. // IN THIS CASE OUR PHASOR
	// Additional mean gradient is accumulated in below methods.
	dL_dmeans[idx] += glm::vec3(dL_dmean.x, dL_dmean.y, dL_dmean.z);
}

// Backward version of INVERSE 2D covariance matrix computation
// (due to length launched as separate kernel before other 
// backward steps contained in preprocess)
__global__ void computeCov2DCUDA(int P,
	const float3* means,
	const int* radii,
	const float* cov3Ds,
	const float h_x, float h_y,
	const float tan_fovx, float tan_fovy,
	const float* view_matrix,
	const float* dL_dconics,
	float3* dL_dmeans,
	float* dL_dcov)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	// Reading location of 3D covariance for this Gaussian
	const float* cov3D = cov3Ds + 6 * idx;

	// Fetch gradients, recompute 2D covariance and relevant 
	// intermediate forward results needed in the backward.
	float3 mean = means[idx];
	float3 dL_dconic = { dL_dconics[4 * idx], dL_dconics[4 * idx + 1], dL_dconics[4 * idx + 3] };
	float3 t = transformPoint4x3(mean, view_matrix);
	
	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;
	
	const float x_grad_mul = txtz < -limx || txtz > limx ? 0 : 1;
	const float y_grad_mul = tytz < -limy || tytz > limy ? 0 : 1;

	glm::mat3 J = glm::mat3(h_x / t.z, 0.0f, -(h_x * t.x) / (t.z * t.z),
		0.0f, h_y / t.z, -(h_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::mat3 W = glm::mat3(
		view_matrix[0], view_matrix[4], view_matrix[8],
		view_matrix[1], view_matrix[5], view_matrix[9],
		view_matrix[2], view_matrix[6], view_matrix[10]);

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 T = W * J;

	glm::mat3 cov2D = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Use helper variables for 2D covariance entries. More compact.
	float a = cov2D[0][0] += 0.3f;
	float b = cov2D[0][1];
	float c = cov2D[1][1] += 0.3f;

	float denom = a * c - b * b;
	float dL_da = 0, dL_db = 0, dL_dc = 0;
	float denom2inv = 1.0f / ((denom * denom) + 0.0000001f);

	if (denom2inv != 0)
	{
		// Gradients of loss w.r.t. entries of 2D covariance matrix,
		// given gradients of loss w.r.t. conic matrix (inverse covariance matrix).
		// e.g., dL / da = dL / d_conic_a * d_conic_a / d_a
		dL_da = denom2inv * (-c * c * dL_dconic.x + 2 * b * c * dL_dconic.y + (denom - a * c) * dL_dconic.z);
		dL_dc = denom2inv * (-a * a * dL_dconic.z + 2 * a * b * dL_dconic.y + (denom - a * c) * dL_dconic.x);
		dL_db = denom2inv * 2 * (b * c * dL_dconic.x - (denom + 2 * b * b) * dL_dconic.y + a * b * dL_dconic.z);

		// Gradients of loss L w.r.t. each 3D covariance matrix (Vrk) entry, 
		// given gradients w.r.t. 2D covariance matrix (diagonal).
		// cov2D = transpose(T) * transpose(Vrk) * T;
		dL_dcov[6 * idx + 0] = (T[0][0] * T[0][0] * dL_da + T[0][0] * T[1][0] * dL_db + T[1][0] * T[1][0] * dL_dc);
		dL_dcov[6 * idx + 3] = (T[0][1] * T[0][1] * dL_da + T[0][1] * T[1][1] * dL_db + T[1][1] * T[1][1] * dL_dc);
		dL_dcov[6 * idx + 5] = (T[0][2] * T[0][2] * dL_da + T[0][2] * T[1][2] * dL_db + T[1][2] * T[1][2] * dL_dc);

		// Gradients of loss L w.r.t. each 3D covariance matrix (Vrk) entry, 
		// given gradients w.r.t. 2D covariance matrix (off-diagonal).
		// Off-diagonal elements appear twice --> double the gradient.
		// cov2D = transpose(T) * transpose(Vrk) * T;
		dL_dcov[6 * idx + 1] = 2 * T[0][0] * T[0][1] * dL_da + (T[0][0] * T[1][1] + T[0][1] * T[1][0]) * dL_db + 2 * T[1][0] * T[1][1] * dL_dc;
		dL_dcov[6 * idx + 2] = 2 * T[0][0] * T[0][2] * dL_da + (T[0][0] * T[1][2] + T[0][2] * T[1][0]) * dL_db + 2 * T[1][0] * T[1][2] * dL_dc;
		dL_dcov[6 * idx + 4] = 2 * T[0][2] * T[0][1] * dL_da + (T[0][1] * T[1][2] + T[0][2] * T[1][1]) * dL_db + 2 * T[1][1] * T[1][2] * dL_dc;
	}
	else
	{
		for (int i = 0; i < 6; i++)
			dL_dcov[6 * idx + i] = 0;
	}

	// Gradients of loss w.r.t. upper 2x3 portion of intermediate matrix T
	// cov2D = transpose(T) * transpose(Vrk) * T;
	float dL_dT00 = 2 * (T[0][0] * Vrk[0][0] + T[0][1] * Vrk[0][1] + T[0][2] * Vrk[0][2]) * dL_da +
		(T[1][0] * Vrk[0][0] + T[1][1] * Vrk[0][1] + T[1][2] * Vrk[0][2]) * dL_db;
	float dL_dT01 = 2 * (T[0][0] * Vrk[1][0] + T[0][1] * Vrk[1][1] + T[0][2] * Vrk[1][2]) * dL_da +
		(T[1][0] * Vrk[1][0] + T[1][1] * Vrk[1][1] + T[1][2] * Vrk[1][2]) * dL_db;
	float dL_dT02 = 2 * (T[0][0] * Vrk[2][0] + T[0][1] * Vrk[2][1] + T[0][2] * Vrk[2][2]) * dL_da +
		(T[1][0] * Vrk[2][0] + T[1][1] * Vrk[2][1] + T[1][2] * Vrk[2][2]) * dL_db;
	float dL_dT10 = 2 * (T[1][0] * Vrk[0][0] + T[1][1] * Vrk[0][1] + T[1][2] * Vrk[0][2]) * dL_dc +
		(T[0][0] * Vrk[0][0] + T[0][1] * Vrk[0][1] + T[0][2] * Vrk[0][2]) * dL_db;
	float dL_dT11 = 2 * (T[1][0] * Vrk[1][0] + T[1][1] * Vrk[1][1] + T[1][2] * Vrk[1][2]) * dL_dc +
		(T[0][0] * Vrk[1][0] + T[0][1] * Vrk[1][1] + T[0][2] * Vrk[1][2]) * dL_db;
	float dL_dT12 = 2 * (T[1][0] * Vrk[2][0] + T[1][1] * Vrk[2][1] + T[1][2] * Vrk[2][2]) * dL_dc +
		(T[0][0] * Vrk[2][0] + T[0][1] * Vrk[2][1] + T[0][2] * Vrk[2][2]) * dL_db;

	// Gradients of loss w.r.t. upper 3x2 non-zero entries of Jacobian matrix
	// T = W * J
	float dL_dJ00 = W[0][0] * dL_dT00 + W[0][1] * dL_dT01 + W[0][2] * dL_dT02;
	float dL_dJ02 = W[2][0] * dL_dT00 + W[2][1] * dL_dT01 + W[2][2] * dL_dT02;
	float dL_dJ11 = W[1][0] * dL_dT10 + W[1][1] * dL_dT11 + W[1][2] * dL_dT12;
	float dL_dJ12 = W[2][0] * dL_dT10 + W[2][1] * dL_dT11 + W[2][2] * dL_dT12;

	float tz = 1.f / t.z;
	float tz2 = tz * tz;
	float tz3 = tz2 * tz;

	// Gradients of loss w.r.t. transformed Gaussian mean t
	float dL_dtx = x_grad_mul * -h_x * tz2 * dL_dJ02;
	float dL_dty = y_grad_mul * -h_y * tz2 * dL_dJ12;
	float dL_dtz = -h_x * tz2 * dL_dJ00 - h_y * tz2 * dL_dJ11 + (2 * h_x * t.x) * tz3 * dL_dJ02 + (2 * h_y * t.y) * tz3 * dL_dJ12;

	// Account for transformation of mean to t
	// t = transformPoint4x3(mean, view_matrix);
	float3 dL_dmean = transformVec4x3Transpose({ dL_dtx, dL_dty, dL_dtz }, view_matrix);

	// Gradients of loss w.r.t. Gaussian means, but only the portion 
	// that is caused because the mean affects the covariance matrix.
	// Additional mean gradient is accumulated in BACKWARD::preprocess.
	dL_dmeans[idx] = dL_dmean;
}

// Backward pass for the conversion of scale and rotation to a 
// 3D covariance matrix for each Gaussian. 
__device__ void computeCov3D(int idx, const glm::vec3 scale, float mod, const glm::vec4 rot, const float* dL_dcov3Ds, glm::vec3* dL_dscales, glm::vec4* dL_drots)
{
	// Recompute (intermediate) results for the 3D covariance computation.
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 S = glm::mat3(1.0f);

	glm::vec3 s = mod * scale;
	S[0][0] = s.x;
	S[1][1] = s.y;
	S[2][2] = s.z;

	glm::mat3 M = S * R;

	const float* dL_dcov3D = dL_dcov3Ds + 6 * idx;

	glm::vec3 dunc(dL_dcov3D[0], dL_dcov3D[3], dL_dcov3D[5]);
	glm::vec3 ounc = 0.5f * glm::vec3(dL_dcov3D[1], dL_dcov3D[2], dL_dcov3D[4]);

	// Convert per-element covariance loss gradients to matrix form
	glm::mat3 dL_dSigma = glm::mat3(
		dL_dcov3D[0], 0.5f * dL_dcov3D[1], 0.5f * dL_dcov3D[2],
		0.5f * dL_dcov3D[1], dL_dcov3D[3], 0.5f * dL_dcov3D[4],
		0.5f * dL_dcov3D[2], 0.5f * dL_dcov3D[4], dL_dcov3D[5]
	);

	// Compute loss gradient w.r.t. matrix M
	// dSigma_dM = 2 * M
	glm::mat3 dL_dM = 2.0f * M * dL_dSigma;

	glm::mat3 Rt = glm::transpose(R);
	glm::mat3 dL_dMt = glm::transpose(dL_dM);

	// Gradients of loss w.r.t. scale
	glm::vec3* dL_dscale = dL_dscales + idx;
	dL_dscale->x = glm::dot(Rt[0], dL_dMt[0]);
	dL_dscale->y = glm::dot(Rt[1], dL_dMt[1]);
	dL_dscale->z = glm::dot(Rt[2], dL_dMt[2]);

	dL_dMt[0] *= s.x;
	dL_dMt[1] *= s.y;
	dL_dMt[2] *= s.z;

	// Gradients of loss w.r.t. normalized quaternion
	glm::vec4 dL_dq;
	dL_dq.x = 2 * z * (dL_dMt[0][1] - dL_dMt[1][0]) + 2 * y * (dL_dMt[2][0] - dL_dMt[0][2]) + 2 * x * (dL_dMt[1][2] - dL_dMt[2][1]);
	dL_dq.y = 2 * y * (dL_dMt[1][0] + dL_dMt[0][1]) + 2 * z * (dL_dMt[2][0] + dL_dMt[0][2]) + 2 * r * (dL_dMt[1][2] - dL_dMt[2][1]) - 4 * x * (dL_dMt[2][2] + dL_dMt[1][1]);
	dL_dq.z = 2 * x * (dL_dMt[1][0] + dL_dMt[0][1]) + 2 * r * (dL_dMt[2][0] - dL_dMt[0][2]) + 2 * z * (dL_dMt[1][2] + dL_dMt[2][1]) - 4 * y * (dL_dMt[2][2] + dL_dMt[0][0]);
	dL_dq.w = 2 * r * (dL_dMt[0][1] - dL_dMt[1][0]) + 2 * x * (dL_dMt[2][0] + dL_dMt[0][2]) + 2 * y * (dL_dMt[1][2] + dL_dMt[2][1]) - 4 * z * (dL_dMt[1][1] + dL_dMt[0][0]);

	// Gradients of loss w.r.t. unnormalized quaternion
	float4* dL_drot = (float4*)(dL_drots + idx);
	*dL_drot = float4{ dL_dq.x, dL_dq.y, dL_dq.z, dL_dq.w };//dnormvdv(float4{ rot.x, rot.y, rot.z, rot.w }, float4{ dL_dq.x, dL_dq.y, dL_dq.z, dL_dq.w });
}

// Backward pass of the preprocessing steps, except
// for the covariance computation and inversion
// (those are handled by a previous kernel call)
template<int C, int PH, int CW>
__global__ void preprocessCUDA(
	int P, int D, int M, int M_p,
	const float3* means,
	const int* radii,
	const float* shs, const float* shs_p,
	const bool* clamped, const bool* clamped_p,
	const glm::vec3* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* view,
	const float* proj,
	const glm::vec3* campos,
	float3* dL_dmean2D,
	glm::vec3* dL_dmeans,
	float* dL_dcolor, float* dL_dphasor, float* dL_ddist_to_light, float* dL_ddist_to_light_ndc,
	float* dL_dcov3D,
	float* dL_dsh, float* dL_dsh_p,
	glm::vec3* dL_dscale,
	glm::vec4* dL_drot, float* dL_dphase_offset, float* dL_ddc_offset,
	const float* pa_ptr, const float* dists_to_light,
	float near_n, float far_n, float dist2phase, bool use_view_dependent_phase, float phase_offset, float dc_offset
	)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	float3 m = means[idx];

	// Taking care of gradients from the screenspace points
	float4 m_hom = transformPoint4x4(m, proj);
	float m_w = 1.0f / (m_hom.w + 0.0000001f);

	float3 m_view = transformPoint4x3(m, view);

	// Compute loss gradient w.r.t. 3D means due to gradients of 2D means
	// from rendering procedure
	glm::vec3 dL_dmean;
	float mul1 = (proj[0] * m.x + proj[4] * m.y + proj[8] * m.z + proj[12]) * m_w * m_w;
	float mul2 = (proj[1] * m.x + proj[5] * m.y + proj[9] * m.z + proj[13]) * m_w * m_w;
	dL_dmean.x = (proj[0] * m_w - proj[3] * mul1) * dL_dmean2D[idx].x + (proj[1] * m_w - proj[3] * mul2) * dL_dmean2D[idx].y;
	dL_dmean.y = (proj[4] * m_w - proj[7] * mul1) * dL_dmean2D[idx].x + (proj[5] * m_w - proj[7] * mul2) * dL_dmean2D[idx].y;
	dL_dmean.z = (proj[8] * m_w - proj[11] * mul1) * dL_dmean2D[idx].x + (proj[9] * m_w - proj[11] * mul2) * dL_dmean2D[idx].y;

	// float mul3 = (proj[2] * m.x + proj[6] * m.y + proj[10] * m.z + proj[14]) * m_w * m_w;
	// dL_dmean.x += (proj[2] * m_w - proj[3] * mul3) * dL_dz_ndc[idx];
	// dL_dmean.y += (proj[6] * m_w - proj[7] * mul3) * dL_dz_ndc[idx];
	// dL_dmean.z += (proj[10] * m_w - proj[11] * mul3) * dL_dz_ndc[idx];

	// That's the second part of the mean gradient. Previous computation
	// of cov2D and following SH conversion also affects it.
	dL_dmeans[idx] += dL_dmean;

	// Compute gradient updates due to computing colors from SHs
	if ((shs != nullptr))
		computeColorFromSH(idx, D, M, (glm::vec3*)means, *campos, shs, clamped, (glm::vec3*)dL_dcolor, (glm::vec3*)dL_dmeans, (glm::vec3*)dL_dsh);

	float dist_to_light = dists_to_light[idx];
	// Compute gradient updates due to computing phasors from SHs
	if ((shs_p != nullptr)) 
	{
		float dL_dCWToF[CW] = { 0 };

		float phase = dist_to_light * dist2phase + phase_offset;
		if (use_view_dependent_phase) phase += pa_ptr[idx * CW + 0];
		float amplitude = pa_ptr[idx * CW + 1];
		float factor = 1.0f / (dist_to_light * dist_to_light);

		float dL_dR = dL_dphasor[idx * PH + 0];
		float dL_dI = dL_dphasor[idx * PH + 1];
		float dL_dA = dL_dphasor[idx * PH + 2];
		// Quad
		float dL_dq1 = dL_dphasor[idx * PH + 3];
		float dL_dq2 = dL_dphasor[idx * PH + 4];
		float dL_dq3 = dL_dphasor[idx * PH + 5];
		float dL_dq4 = dL_dphasor[idx * PH + 6];

		float sin_p = sinf(phase);
		float cos_p = cosf(phase);

		// Gradient of phase from sh
		if (use_view_dependent_phase)
		{
			dL_dCWToF[0] = (
				dL_dR * -sin_p + dL_dI * cos_p + // R & I
				dL_dq1 * -sin_p + dL_dq2 * sin_p + dL_dq3 * cos_p + dL_dq4 * -cos_p // quad
				) * amplitude * factor;
		}
		atomicAdd(&(dL_dphase_offset[0]), (
			dL_dR * -sin_p + dL_dI * cos_p + 
			dL_dq1 * -sin_p + dL_dq2 * sin_p + dL_dq3 * cos_p + dL_dq4 * -cos_p
			) * amplitude * factor);
		
		// Gradient of amplitude from sh
		dL_dCWToF[1] = (
			dL_dR * cos_p + dL_dI * sin_p + dL_dA + // R & I
			dL_dq1 * (cos_p + dc_offset) + dL_dq2 * (-cos_p + dc_offset) + 
			dL_dq3 * (sin_p + dc_offset) + dL_dq4 * (-sin_p + dc_offset)
			) * factor;
		atomicAdd(&(dL_ddc_offset[0]), (dL_dq1 + dL_dq2 + dL_dq3 + dL_dq4) * amplitude * factor);

		// Gradient of means3D
		float coeff = (
			dL_dR * -sin_p + dL_dI *  cos_p + 
			dL_dq1 * -sin_p + dL_dq2 * sin_p + dL_dq3 * cos_p + dL_dq4 * -cos_p
			) * dist2phase * amplitude * factor / dist_to_light + (
				dL_dR * -cos_p + dL_dI * -sin_p - dL_dA +
				dL_dq1 * -(cos_p + dc_offset) + dL_dq2 * (cos_p - dc_offset) + 
				dL_dq3 * -(sin_p + dc_offset) + dL_dq4 * (sin_p - dc_offset)
			) * 2.0f * amplitude * factor * factor;
		float dL_dx_view = m_view.x * coeff;
		float dL_dy_view = m_view.y * coeff;
		float dL_dz_view = m_view.z * coeff;
		float dL_dx = dL_dx_view * view[0] + dL_dy_view * view[1] + dL_dz_view * view[2];
		float dL_dy = dL_dx_view * view[4] + dL_dy_view * view[5] + dL_dz_view * view[6];
		float dL_dz = dL_dx_view * view[8] + dL_dy_view * view[9] + dL_dz_view * view[10];
		dL_dmeans[idx] += glm::vec3(dL_dx, dL_dy, dL_dz);

		computePhasorFromSH(idx, D, M_p, (glm::vec3*)means, *campos, shs_p, clamped_p, (glm::vec2*)dL_dCWToF, (glm::vec3*)dL_dmeans, (glm::vec2*)dL_dsh_p);
	}

	float dndc_dist_ddist = (far_n * near_n) / ((far_n - near_n) * dist_to_light * dist_to_light);
	float dL_ddist = dL_ddist_to_light_ndc[idx] * dndc_dist_ddist + dL_ddist_to_light[idx];

	// dL_dmean2D[idx].z += dL_ddist_to_light_ndc[idx] * dndc_dist_ddist;

	float dL_dx_view = dL_ddist * m_view.x / dist_to_light;
	float dL_dy_view = dL_ddist * m_view.y / dist_to_light;
	float dL_dz_view = dL_ddist * m_view.z / dist_to_light;
	
	float dL_dx = dL_dx_view * view[0] + dL_dy_view * view[1] + dL_dz_view * view[2];
	float dL_dy = dL_dx_view * view[4] + dL_dy_view * view[5] + dL_dz_view * view[6];
	float dL_dz = dL_dx_view * view[8] + dL_dy_view * view[9] + dL_dz_view * view[10];
	dL_dmeans[idx] += glm::vec3(dL_dx, dL_dy, dL_dz);

	// Compute gradient updates due to computing covariance from scale/rotation
	if (scales)
		computeCov3D(idx, scales[idx], scale_modifier, rotations[idx], dL_dcov3D, dL_dscale, dL_drot);
}

// Backward version of the rendering procedure.
template <uint32_t C, uint32_t P>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float* __restrict__ bg_color,
	const float2* __restrict__ points_xy_image,
	const float4* __restrict__ conic_opacity,
	const float* __restrict__ colors, const float* __restrict__ phasors, const float* __restrict__ dists_to_light, const float* __restrict__ dists_to_light_ndc,
	const float* __restrict__ final_Ts,
	const float* __restrict__ alpha_totals,
	const float* __restrict__ w_z_totals, const float* __restrict__ w_z2_totals,
	const float* __restrict__ w_amplitude_totals, const float* __restrict__ w_amplitude2_totals,
	const uint32_t* __restrict__ n_contrib,
	const float* __restrict__ dL_dpixels, const float* __restrict__ dL_dpixels_p, 
	const float* __restrict__ dL_dpixels_d, const float* __restrict__ dL_dpixels_a, 
	const float* __restrict__ dL_dpixels_e, const float* __restrict__ dL_dpixels_dd, const float* __restrict__ dL_dpixels_ad,
	float3* __restrict__ dL_dmean2D,
	float4* __restrict__ dL_dconic2D,
	float* __restrict__ dL_dopacity,
	float* __restrict__ dL_dcolors, float* __restrict__ dL_dphasors, float* __restrict__ dL_ddists_to_light, float* __restrict__ dL_ddists_to_light_ndc)
{
	// We rasterize again. Compute necessary block info.
	auto block = cg::this_thread_block();
	const uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	const uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	const uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	const uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	const uint32_t pix_id = W * pix.y + pix.x;
	const float2 pixf = { (float)pix.x, (float)pix.y };

	const bool inside = pix.x < W&& pix.y < H;
	const uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];

	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);

	bool done = !inside;
	int toDo = range.y - range.x;

	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];
	__shared__ float collected_colors[C * BLOCK_SIZE];
	__shared__ float collected_phasors[P * BLOCK_SIZE];
	__shared__ float collected_dists_to_light[BLOCK_SIZE];
	__shared__ float collected_dists_to_light_ndc[BLOCK_SIZE];

	// In the forward, we stored the final value for T, the
	// product of all (1 - alpha) factors. 
	const float T_final = inside ? final_Ts[pix_id] : 0;
	float T = T_final;

	// We start from the back. The ID of the last contributing
	// Gaussian is known from each pixel from the forward.
	uint32_t contributor = toDo;
	const int last_contributor = inside ? n_contrib[pix_id] : 0;

	const float alpha_total = inside ? alpha_totals[pix_id] : 0;
	const float w_z_total = inside ? w_z_totals[pix_id] : 0;
	const float w_z2_total = inside ? w_z2_totals[pix_id] : 0;
	// const float w_amplitude_total = inside ? w_amplitude_totals[pix_id] : 0;
	// const float w_amplitude2_total = inside ? w_amplitude2_totals[pix_id] : 0;

	float accum_rec[C] = { 0 };
	float dL_dpixel[C];
	float accum_rec_p[P] = { 0 };
	float dL_dpixel_p[P];
	float accum_rec_d = 0;
	float dL_dpixel_d;
	float accum_rec_a = 0;
	float dL_dpixel_a;
	// float dL_dpixel_e;
	float accum_rec_dd = 0;
	float dL_dpixel_dd;
	// float accum_rec_ad = 0;
	// float dL_dpixel_ad;
	if (inside)
	{
		for (int i = 0; i < C; i++)
			dL_dpixel[i] = dL_dpixels[i * H * W + pix_id];
		for (int i = 0; i < P; i++)
			dL_dpixel_p[i] = dL_dpixels_p[i * H * W + pix_id];
		dL_dpixel_d = dL_dpixels_d[pix_id];
		dL_dpixel_a = dL_dpixels_a[pix_id];
		// dL_dpixel_e = dL_dpixels_e[pix_id];
		dL_dpixel_dd = dL_dpixels_dd[pix_id];
		// dL_dpixel_ad = dL_dpixels_ad[pix_id];
	}

	float last_alpha = 0;
	float last_color[C] = { 0 };
	float last_phasor[P] = { 0 };
	float last_dist = { 0 };
	float last_dL_dw = { 0 };
	// float last_dL_dw_a = { 0 };

	// Gradient of pixel coordinate w.r.t. normalized 
	// screen-space viewport corrdinates (-1 to 1)
	const float ddelx_dx = 0.5 * W;
	const float ddely_dy = 0.5 * H;

	// Traverse all Gaussians
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// Load auxiliary data into shared memory, start in the BACK
		// and load them in revers order.
		block.sync();
		const int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			const int coll_id = point_list[range.y - progress - 1];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];

			for (int i = 0; i < C; i++)
				collected_colors[i * BLOCK_SIZE + block.thread_rank()] = colors[coll_id * C + i];
			for (int i = 0; i < P; i++)
				collected_phasors[i * BLOCK_SIZE + block.thread_rank()] = phasors[coll_id * P + i];
			collected_dists_to_light[block.thread_rank()] = dists_to_light[coll_id];
			collected_dists_to_light_ndc[block.thread_rank()] = dists_to_light_ndc[coll_id];
		}
		block.sync();

		// Iterate over Gaussians
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current Gaussian ID. Skip, if this one
			// is behind the last contributor for this pixel.
			contributor--;
			if (contributor >= last_contributor)
				continue;

			// Compute blending values, as before.
			const float2 xy = collected_xy[j];
			const float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			const float4 con_o = collected_conic_opacity[j];
			const float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			const float G = exp(power);
			const float alpha = min(0.99f, con_o.w * G);
			if (alpha < 1.0f / 255.0f)
				continue;

			T = T / (1.f - alpha);
			const float dchannel_dcolor = alpha * T;
			const float dchannel_dphasor = alpha * T * T;
			// const float dchannel_dphasor = alpha * T;
			const float dchannel_ddepth = alpha * T;

			// Propagate gradients to per-Gaussian colors and keep
			// gradients w.r.t. alpha (blending factor for a Gaussian/pixel
			// pair).
			float dL_dalpha = 0.0f;
			float dL_dalpha_c = 0.0f;
			float dL_dalpha_p = 0.0f;
			float dL_dalpha_d = 0.0f;
			float dL_dalpha_a = 0.0f;
			// float dL_dalpha_e = 0.0f;
			float dL_dalpha_dd = 0.0f;
			// float dL_dalpha_ad = 0.0f;
			const int global_id = collected_id[j];

			// Color
			for (int ch = 0; ch < C; ch++)
			{
				const float c = collected_colors[ch * BLOCK_SIZE + j];
				// Update last color (to be used in the next iteration)
				accum_rec[ch] = last_alpha * last_color[ch] + (1.f - last_alpha) * accum_rec[ch];
				last_color[ch] = c;

				const float dL_dchannel = dL_dpixel[ch];
				dL_dalpha_c += (c - accum_rec[ch]) * dL_dchannel;
				// Update the gradients w.r.t. color of the Gaussian. 
				// Atomic, since this pixel is just one of potentially
				// many that were affected by this Gaussian.
				atomicAdd(&(dL_dcolors[global_id * C + ch]), dchannel_dcolor * dL_dchannel);
			}
			dL_dalpha_c *= T;

			// ToF
			for (int ch = 0; ch < P; ch++)
			{
				const float p = collected_phasors[ch * BLOCK_SIZE + j];
				accum_rec_p[ch] = last_alpha * last_phasor[ch] + (1.f - last_alpha) * (1.f - last_alpha) * accum_rec_p[ch];
				// accum_rec_p[ch] = last_alpha * last_phasor[ch] + (1.f - last_alpha) * accum_rec_p[ch];
				last_phasor[ch] = p;
				const float dL_dchannel_p = dL_dpixel_p[ch];
				dL_dalpha_p += (p - 2.f * (1.f - alpha) * accum_rec_p[ch]) * dL_dchannel_p;
				// dL_dalpha_p += (p - accum_rec_p[ch]) * dL_dchannel_p;
				atomicAdd(&(dL_dphasors[global_id * P + ch]), dchannel_dphasor * dL_dchannel_p);
			}
			dL_dalpha_p *= T * T;

			// Depth
			const float dist = collected_dists_to_light[j];
			accum_rec_d = last_alpha * last_dist + (1.f - last_alpha) * accum_rec_d;
			last_dist = dist;
			const float dL_dchannel_d = dL_dpixel_d;
			dL_dalpha_d += (dist - accum_rec_d) * dL_dchannel_d;
			atomicAdd(&(dL_ddists_to_light[global_id]), dchannel_ddepth * dL_dchannel_d);
			dL_dalpha_d *= T;

			// Acc
			accum_rec_a = last_alpha + (1.f - last_alpha) * accum_rec_a;
			dL_dalpha_a += (1.f - accum_rec_a) * dL_dpixel_a;
			dL_dalpha_a *= T;

			// // Entropy
			// if (alpha_total >= 1.0f / 255.0f)
			// 	dL_dalpha_e = dL_dpixel_e * -(log(alpha / alpha_total) + 1) * (alpha_total - alpha) / (alpha_total * alpha_total);

			// Depth distortion
			const float z = collected_dists_to_light_ndc[j];
			// dL_dalpha_dd = dL_dpixel_dd * 2.0f * (z * z * alpha_total - 2.0f * z * w_z_total + w_z2_total);
			// atomicAdd(&(dL_ddists_to_light_ndc[global_id]), dL_dpixel_dd * 2.0f * alpha * T * (z * alpha_total - w_z_total));
			float dL_dw = dL_dpixel_dd * (z * z * (1 - T_final) - 2.0f * z * w_z_total + w_z2_total);
			accum_rec_dd = last_alpha * last_dL_dw + (1.f - last_alpha) * accum_rec_dd;
			last_dL_dw = dL_dw;
			dL_dalpha_dd += dL_dw - accum_rec_dd;
			atomicAdd(&(dL_ddists_to_light_ndc[global_id]), dL_dpixel_dd * 2.0f * alpha * T * (z * (1 - T_final) - w_z_total));
			dL_dalpha_dd *= T;

			// // Amplitude distortion
			// const float amplitude = collected_phasors[2 * BLOCK_SIZE + j] * collected_dists_to_light[j] * collected_dists_to_light[j];
			// float dL_dw_a = 0;//dL_dpixel_ad * (amplitude * amplitude * (1 - T_final) - 2.0f * amplitude * w_amplitude_total + w_amplitude2_total);
			// accum_rec_ad = last_alpha * last_dL_dw_a + (1.f - last_alpha) * accum_rec_ad;
			// last_dL_dw_a = dL_dw_a;
			// dL_dalpha_ad += dL_dw_a - accum_rec_ad;
			// atomicAdd(&(dL_dphasors[global_id * 3 + 2]), dL_dpixel_ad * 2.0f * alpha * T * (T * T * amplitude * (1 - T_final) - T * w_amplitude_total));
			// // atomicAdd(&(dL_dphasors[global_id * 3 + 2]), dL_dpixel_ad * 2.0f * alpha * T * (amplitude * (1 - T_final) - w_amplitude_total));
			// dL_dalpha_ad *= T;

			// Update last alpha (to be used in the next iteration)
			last_alpha = alpha;

			// Account for fact that alpha also influences how much of
			// the background color is added if nothing left to blend
			float bg_dot_dpixel = 0;
			for (int i = 0; i < C; i++)
				bg_dot_dpixel += bg_color[i * H * W + pix_id] * dL_dpixel[i];
			dL_dalpha += (-T_final / (1.f - alpha)) * bg_dot_dpixel;

			float bg_dot_dpixel_p = 0;
			for (int i = 0; i < P; i++)
				bg_dot_dpixel_p += bg_color[i * H * W + pix_id] * dL_dpixel_p[i];
			dL_dalpha_p += (-T_final / (1.f - alpha)) * bg_dot_dpixel_p;

			dL_dalpha += dL_dalpha_c;
			dL_dalpha += dL_dalpha_p;
			dL_dalpha += dL_dalpha_d;
			dL_dalpha += dL_dalpha_a;
			// dL_dalpha += dL_dalpha_e;
			dL_dalpha += dL_dalpha_dd;
			// dL_dalpha += dL_dalpha_ad;

			// Helpful reusable temporary variables
			const float dL_dG = con_o.w * dL_dalpha;
			const float gdx = G * d.x;
			const float gdy = G * d.y;
			const float dG_ddelx = -gdx * con_o.x - gdy * con_o.y;
			const float dG_ddely = -gdy * con_o.z - gdx * con_o.y;

			// Update gradients w.r.t. 2D mean position of the Gaussian
			atomicAdd(&dL_dmean2D[global_id].x, dL_dG * dG_ddelx * ddelx_dx);
			atomicAdd(&dL_dmean2D[global_id].y, dL_dG * dG_ddely * ddely_dy);
			// atomicAdd(&dL_dmean2D[global_id].z, dL_ddists_to_light_ndc[global_id]);

			// Update gradients w.r.t. 2D covariance (2x2 matrix, symmetric)
			atomicAdd(&dL_dconic2D[global_id].x, -0.5f * gdx * d.x * dL_dG);
			atomicAdd(&dL_dconic2D[global_id].y, -0.5f * gdx * d.y * dL_dG);
			atomicAdd(&dL_dconic2D[global_id].w, -0.5f * gdy * d.y * dL_dG);

			// Update gradients w.r.t. opacity of the Gaussian
			atomicAdd(&(dL_dopacity[global_id]), G * dL_dalpha);
		}
	}
}

void BACKWARD::preprocess(
	int P, int D, int M, int M_p,
	const float3* means3D,
	const int* radii,
	const float* shs, const float* shs_p,
	const bool* clamped, const bool* clamped_p,
	const glm::vec3* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* cov3Ds,
	const float* viewmatrix,
	const float* projmatrix,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	const glm::vec3* campos,
	float3* dL_dmean2D,
	const float* dL_dconic,
	glm::vec3* dL_dmean3D,
	float* dL_dcolor, float* dL_dphasor, float* dL_ddist_to_light, float* dL_ddist_to_light_ndc,
	float* dL_dcov3D,
	float* dL_dsh, float* dL_dsh_p,
	glm::vec3* dL_dscale,
	glm::vec4* dL_drot, float* dL_dphase_offset, float* dL_ddc_offset,
	const float* phase_amplitude_from_sh_ptr, const float* dists_to_light_ptr,
	float near_n, float far_n, float depth_range, bool use_view_dependent_phase, float phase_offset, float dc_offset
	)
{
	// Propagate gradients for the path of 2D conic matrix computation. 
	// Somewhat long, thus it is its own kernel rather than being part of 
	// "preprocess". When done, loss gradient w.r.t. 3D means has been
	// modified and gradient w.r.t. 3D covariance matrix has been computed.	
	computeCov2DCUDA << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		radii,
		cov3Ds,
		focal_x,
		focal_y,
		tan_fovx,
		tan_fovy,
		viewmatrix,
		dL_dconic,
		(float3*)dL_dmean3D,
		dL_dcov3D);

	const float dist2phase = 4.0f * PI / depth_range;

	// Propagate gradients for remaining steps: finish 3D mean gradients,
	// propagate color gradients to SH (if desireD), propagate 3D covariance
	// matrix gradients to scale and rotation.
	preprocessCUDA<NUM_CHANNELS, NUM_CHANNELS_PHASOR, NUM_CHANNELS_CWTOF> << < (P + 255) / 256, 256 >> > (
		P, D, M, M_p,
		(float3*)means3D,
		radii,
		shs, shs_p,
		clamped, clamped_p,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		viewmatrix,
		projmatrix,
		campos,
		(float3*)dL_dmean2D,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor, dL_dphasor, dL_ddist_to_light, dL_ddist_to_light_ndc,
		dL_dcov3D,
		dL_dsh, dL_dsh_p,
		dL_dscale,
		dL_drot, dL_dphase_offset, dL_ddc_offset,
		phase_amplitude_from_sh_ptr, dists_to_light_ptr,
		near_n, far_n, dist2phase, use_view_dependent_phase, phase_offset, dc_offset
		);
}

void BACKWARD::render(
	const dim3 grid, const dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float* bg_color,
	const float2* means2D,
	const float4* conic_opacity,
	const float* colors, const float* phasors, const float* dists_to_light, const float* dists_to_light_ndc,
	const float* final_Ts,
	const float* alpha_totals,
	const float* w_z_total, const float* w_z2_total,
	const float* w_amplitude_total, const float* w_amplitude2_total,
	const uint32_t* n_contrib,
	const float* dL_dpixels, const float* dL_dpixels_p, 
	const float* dL_dpixels_d, const float* dL_dpixels_a, 
	const float* dL_dpixels_e, const float* dL_dpixels_dd, const float* dL_dpixels_ad,
	float3* dL_dmean2D,
	float4* dL_dconic2D,
	float* dL_dopacity,
	float* dL_dcolors, float* dL_dphasors, float* dL_ddists_to_light, float* dL_ddists_to_light_ndc)
{
	renderCUDA<NUM_CHANNELS, NUM_CHANNELS_PHASOR> << <grid, block >> >(
		ranges,
		point_list,
		W, H,
		bg_color,
		means2D,
		conic_opacity,
		colors, phasors, dists_to_light, dists_to_light_ndc,
		final_Ts,
		alpha_totals,
		w_z_total, w_z2_total,
		w_amplitude_total, w_amplitude2_total,
		n_contrib,
		dL_dpixels, dL_dpixels_p, 
		dL_dpixels_d, dL_dpixels_a, 
		dL_dpixels_e, dL_dpixels_dd, dL_dpixels_ad,
		dL_dmean2D,
		dL_dconic2D,
		dL_dopacity,
		dL_dcolors, dL_dphasors, dL_ddists_to_light, dL_ddists_to_light_ndc);
}